#include <sstream>
#include <iostream>
#include <stdexcept>
#include <hip/hip_runtime.h>
#include <multem/multem_ext.h>
#include <types.cuh>
#include <input_multislice.cuh>
#include <device_functions.cuh>
#include <multem.cu>

bool is_gpu_available_wrapper() {
  return mt::is_gpu_available();
}

int number_of_gpu_available_wrapper() {
  return mt::number_of_gpu_available();
}

mt::eDevice string_to_device_enum(const std::string &device) {
  if (device == "host") {
    return mt::e_host;
  } else if (device == "device") {
    return mt::e_device;
  } else if (device == "host_device") {
    return mt::e_host_device;
  }
  throw std::runtime_error("Unknown device");
}

mt::ePrecision string_to_precision_enum(const std::string &precision) {
  if (precision == "float") {
    return mt::eP_float;
  } else if (precision == "double") {
    return mt::eP_double;
  }
  throw std::runtime_error("Unknown precision");
}

mt::eElec_Spec_Int_Model string_to_interation_model_enum(const std::string &interaction_model) {
  if (interaction_model == "Multislice") {
    return mt::eESIM_Multislice;
  } else if (interaction_model == "Phase_Object") {
    return mt::eESIM_Phase_Object;
  } else if (interaction_model == "Weak_Phase_Object") {
    return mt::eESIM_Weak_Phase_Object;
  }
  throw std::runtime_error("Invalid interaction model");
}

mt::ePotential_Type string_to_potential_type_enum(const std::string &potential_type) {
  if (potential_type == "Doyle_0_4") {
    return mt::ePT_Doyle_0_4;
  } else if (potential_type == "Peng_0_4") {
    return mt::ePT_Peng_0_4;
  } else if (potential_type == "Peng_0_12") {
    return mt::ePT_Peng_0_12;
  } else if (potential_type == "Kirkland_0_12") {
    return mt::ePT_Kirkland_0_12;
  } else if (potential_type == "Weickenmeier_0_12") {
    return mt::ePT_Weickenmeier_0_12;
  } else if (potential_type == "Lobato_0_12") {
    return mt::ePT_Lobato_0_12;
  } else if (potential_type == "none") {
    return mt::ePT_none;
  }
  throw std::runtime_error("Invalid potential type");
}

mt::ePhonon_Model string_to_phonon_model_enum(const std::string &pn_model) {
  if (pn_model == "Still_Atom") {
    return mt::ePM_Still_Atom;
  } else if (pn_model == "Absorptive_Model") {
    return mt::ePM_Absorptive_Model;
  } else if (pn_model == "Frozen_Phonon") {
    return mt::ePM_Frozen_Phonon;
  }
  throw std::runtime_error("Invalid phonon model");
}

mt::eRot_Point_Type string_to_rot_point_type_enum(const std::string &spec_rot_center_type) {
  if (spec_rot_center_type == "geometric_center") {
    return mt::eRPT_geometric_center;
  } else if (spec_rot_center_type == "User_Define") {
    return mt::eRPT_User_Define;
  }
  throw std::runtime_error("Invalid spec rot center type");
}

mt::eThick_Type string_to_thick_type_enum(const std::string &thick_type) {
  if (thick_type == "Whole_Spec") {
    return mt::eTT_Whole_Spec;
  } else if (thick_type == "Through_Thick") {
    return mt::eTT_Through_Thick;
  } else if (thick_type == "Through_Slices") {
    return mt::eTT_Through_Slices;
  }
  throw std::runtime_error("Invalid thickness type");
}

mt::ePotential_Slicing string_to_potential_slicing_enum(const std::string &potential_slicing) {
  if (potential_slicing == "Planes") {
    return mt::ePS_Planes;
  } else if (potential_slicing == "dz_Proj") {
    return mt::ePS_dz_Proj;
  } else if (potential_slicing == "dz_Sub") {
    return mt::ePS_dz_Sub;
  } else if (potential_slicing == "Auto") {
    return mt::ePS_Auto;
  }
  throw std::runtime_error("Invalid potential slicing");
}

mt::eTEM_Sim_Type string_to_tem_sim_type_enum(const std::string &simulation_type) {
  if (simulation_type == "STEM") {
    return mt::eTEMST_STEM ;
  } else if (simulation_type == "ISTEM") {
    return mt::eTEMST_ISTEM ;
  } else if (simulation_type == "CBED") {
    return mt::eTEMST_CBED ;
  } else if (simulation_type == "CBEI") {
    return mt::eTEMST_CBEI ;
  } else if (simulation_type == "ED") {
    return mt::eTEMST_ED ;
  } else if (simulation_type == "HRTEM") {
    return mt::eTEMST_HRTEM ;
  } else if (simulation_type == "PED") {
    return mt::eTEMST_PED ;
  } else if (simulation_type == "HCTEM") {
    return mt::eTEMST_HCTEM ;
  } else if (simulation_type == "EWFS") {
    return mt::eTEMST_EWFS ;
  } else if (simulation_type == "EWRS") {
    return mt::eTEMST_EWRS ;
  } else if (simulation_type == "EELS") {
    return mt::eTEMST_EELS ;
  } else if (simulation_type == "EFTEM") {
    return mt::eTEMST_EFTEM ;
  } else if (simulation_type == "IWFS") {
    return mt::eTEMST_IWFS ;
  } else if (simulation_type == "IWRS") {
    return mt::eTEMST_IWRS ;
  } else if (simulation_type == "PPFS") {
    return mt::eTEMST_PPFS ;
  } else if (simulation_type == "PPRS") {
    return mt::eTEMST_PPRS ;
  } else if (simulation_type == "TFFS") {
    return mt::eTEMST_TFFS ;
  } else if (simulation_type == "TFRS") {
    return mt::eTEMST_TFRS ;
  } else if (simulation_type == "PropFS") {
    return mt::eTEMST_PropFS ;
  } else if (simulation_type == "PropRS") {
    return mt::eTEMST_PropRS ;
  }
  throw std::runtime_error("Invalid simulation type");
}

mt::eIncident_Wave_Type string_to_incident_wave_type_enum(const std::string &iw_type) {
  if (iw_type == "eIWT_Plane_Wave") {
    return mt::eIWT_Plane_Wave;
  } else if (iw_type == "eIWT_Convergent_Wave") { 
    return mt::eIWT_Convergent_Wave;
  } else if (iw_type == "eIWT_User_Define_Wave") { 
    return mt::eIWT_User_Define_Wave;
  } else if (iw_type == "eIWT_Auto") { 
    return mt::eIWT_Auto;
  }
  throw std::runtime_error("Invalid iw type");
}

mt::eIllumination_Model string_to_illumination_model_enum(const std::string &illumination_model) {
  if (illumination_model == "Coherent") {
    return mt::eIM_Coherent;
  } else if (illumination_model == "Partial_Coherent") {
    return mt::eIM_Partial_Coherent;
  } else if (illumination_model == "Trans_Cross_Coef") {
    return mt::eIM_Trans_Cross_Coef;
  } else if (illumination_model == "Full_Integration") {
    return mt::eIM_Full_Integration;
  } else if (illumination_model == "none") {
    return mt::eIM_none;
  }
  throw std::runtime_error("Invalid illumination model");
}
  
mt::eOperation_Mode string_to_operation_model_enum(const std::string &operation_mode) {
  if (operation_mode == "Normal") {
    return mt::eOM_Normal;
  } else if (operation_mode == "Advanced") {
    return mt::eOM_Advanced;
  }
  throw std::runtime_error("Invalid operation mode");
}

mt::eLens_Var_Type string_to_lens_var_type_enum(const std::string &cdl_var_type) {
  if (cdl_var_type == "off") {
    return mt::eLVT_off;
  } else if (cdl_var_type == "m") {
    return mt::eLVT_m;
  } else if (cdl_var_type == "f") {
    return mt::eLVT_f;
  } else if (cdl_var_type == "Cs3") {
    return mt::eLVT_Cs3;
  } else if (cdl_var_type == "Cs5") {
    return mt::eLVT_Cs5;
  } else if (cdl_var_type == "mfa2") {
    return mt::eLVT_mfa2;
  } else if (cdl_var_type == "afa2") {
    return mt::eLVT_afa2;
  } else if (cdl_var_type == "mfa3") {
    return mt::eLVT_mfa3;
  } else if (cdl_var_type == "afa3") {
    return mt::eLVT_afa3;
  } else if (cdl_var_type == "inner_aper_ang") {
    return mt::eLVT_inner_aper_ang;
  } else if (cdl_var_type == "outer_aper_ang") {
    return mt::eLVT_outer_aper_ang;
  }
  throw std::runtime_error("Invalid cdl_var_type");
}


SystemConfigurationWrapper::SystemConfigurationWrapper()
    : handle_(new mt::System_Configuration()) {
  handle_->validate_parameters();
}

SystemConfigurationWrapper::SystemConfigurationWrapper(
    const char *device,
    const char *precision,
    int cpu_ncores,
    int cpu_nthread,
    int gpu_device,
    int gpu_stream)
    : handle_(new mt::System_Configuration()) {
  handle_->device = string_to_device_enum(device);
  handle_->precision = string_to_precision_enum(precision);
  handle_->cpu_ncores = cpu_ncores;
  handle_->cpu_nthread = cpu_nthread;
  handle_->gpu_device = gpu_device;
  handle_->gpu_nstream = gpu_stream;
  handle_->active = true;
  handle_->validate_parameters();
  handle_->set_device();
}

int SystemConfigurationWrapper::get_device() {
  return handle_->get_device();
}

bool SystemConfigurationWrapper::is_host() const {
  return handle_->is_host();
}

bool SystemConfigurationWrapper::is_device() const {
  return handle_->is_device();
}

bool SystemConfigurationWrapper::is_float() const {
  return handle_->is_float();
}

bool SystemConfigurationWrapper::is_double() const {
  return handle_->is_double();
}

bool SystemConfigurationWrapper::is_float_host() const {
  return handle_->is_float_host();
}

bool SystemConfigurationWrapper::is_double_host() const {
  return handle_->is_double_host();
}

bool SystemConfigurationWrapper::is_float_device() const {
  return handle_->is_float_device();
}

bool SystemConfigurationWrapper::is_double_device() const {
  return handle_->is_double_device();
}






InputMultisliceWrapper::InputMultisliceWrapper()
  : handle_(new InputMultisliceType()) {}


void InputMultisliceWrapper::set_system_conf(SystemConfigurationWrapper system_conf) {
  handle_->system_conf = *(system_conf.handle());
}

void InputMultisliceWrapper::set_interaction_model(const char *interaction_model) {
  handle_->interaction_model = string_to_interation_model_enum(interaction_model);
}

void InputMultisliceWrapper::set_potential_type(const char *potential_type) {
  handle_->potential_type = string_to_potential_type_enum(potential_type);
}

void InputMultisliceWrapper::set_pn_model(const char *pn_model) {
  handle_->pn_model = string_to_phonon_model_enum(pn_model);
}

void InputMultisliceWrapper::set_pn_coh_contrib(bool pn_coh_contrib) {
  handle_->pn_coh_contrib = pn_coh_contrib;
}

void InputMultisliceWrapper::set_pn_single_conf(bool pn_single_conf) {
  handle_->pn_single_conf = pn_single_conf;
}

void InputMultisliceWrapper::set_fp_dist(int fp_dist) {
  handle_->fp_dist = fp_dist;
}

void InputMultisliceWrapper::set_pn_seed(int pn_seed) {
  handle_->pn_seed = pn_seed;
}

void InputMultisliceWrapper::set_pn_nconf(int pn_nconf) {
  handle_->pn_nconf = pn_nconf;
}

void InputMultisliceWrapper::set_fp_iconf_0(int fp_iconf_0) {
  handle_->fp_iconf_0 = fp_iconf_0;
}

void InputMultisliceWrapper::set_is_crystal(bool is_crystal) {
  handle_->is_crystal = is_crystal;
}

void InputMultisliceWrapper::set_spec_rot_theta(FloatType spec_rot_theta) {
  handle_->spec_rot_theta = spec_rot_theta;
}
  
void InputMultisliceWrapper::set_spec_rot_center_type(const char *spec_rot_center_type) {
  handle_->spec_rot_center_type = string_to_rot_point_type_enum(spec_rot_center_type);
}
  
void InputMultisliceWrapper::set_thick_type(const char *thick_type) {
  handle_->thick_type = string_to_thick_type_enum(thick_type);
}
  
void InputMultisliceWrapper::set_potential_slicing(const char *potential_slicing) {
  handle_->potential_slicing = string_to_potential_slicing_enum(potential_slicing);
}

void InputMultisliceWrapper::set_simulation_type(const char *simulation_type) {
  handle_->simulation_type = string_to_tem_sim_type_enum(simulation_type);
}

void InputMultisliceWrapper::set_iw_type(const char *iw_type) {
  handle_->iw_type = string_to_incident_wave_type_enum(iw_type);
}
  
void InputMultisliceWrapper::set_illumination_model(const char *illumination_model) {
  handle_->illumination_model = string_to_illumination_model_enum(illumination_model);
}

void InputMultisliceWrapper::set_operation_mode(const char *operation_mode) {
  handle_->operation_mode = string_to_operation_model_enum(operation_mode);
}

void InputMultisliceWrapper::set_slice_storage(bool slice_storage) {
  handle_->slice_storage = slice_storage;
}

void InputMultisliceWrapper::set_mul_sign(int mul_sign) {
  handle_->mul_sign = mul_sign;
}

void InputMultisliceWrapper::set_Vrl(FloatType Vrl) {
  handle_->Vrl = Vrl;
}

void InputMultisliceWrapper::set_nR(int nR) {
  handle_->nR = nR;
}

void InputMultisliceWrapper::set_nrot(int nrot) {
  handle_->nrot = nrot;
}

void InputMultisliceWrapper::set_cdl_var_type(const char *cdl_var_type) {
  handle_->cdl_var_type = string_to_lens_var_type_enum(cdl_var_type);
}
  
void InputMultisliceWrapper::set_islice(int islice) {
  handle_->islice = islice;
}

void InputMultisliceWrapper::set_dp_Shift(bool dp_Shift) {
  handle_->dp_Shift = dp_Shift;
}

void InputMultisliceWrapper::set_E_0(FloatType E_0) {
  handle_->E_0 = E_0;
}

void InputMultisliceWrapper::set_lambda(FloatType lambda) {
  handle_->lambda = lambda;
}

void InputMultisliceWrapper::set_theta(FloatType theta) {
  handle_->theta = theta;
}

void InputMultisliceWrapper::set_phi(FloatType phi) {
  handle_->phi = phi;
}


void InputMultisliceWrapper::assign(InputMultisliceWrapper &input_multislice) {
  handle_->assign(*(input_multislice.handle_));
}
  
void InputMultisliceWrapper::validate_parameters() {
  handle_->validate_parameters();
}
  
void InputMultisliceWrapper::validate_output_area() {
  handle_->validate_output_area();
}

void InputMultisliceWrapper::set_iscan_beam_position() {
  handle_->set_iscan_beam_position();
}

void InputMultisliceWrapper::set_reverse_multislice(bool rm) {
  handle_->set_reverse_multislice(rm);
}

/* InputMultisliceWrapper::FloatType InputMultisliceWrapper::Rx_exp_factor() { */
/*   return handle_->Rx_exp_factor(); */
/* } */

/* InputMultisliceWrapper::FloatType InputMultisliceWrapper::Ry_exp_factor() { */
/*   return handle_->Ry_exp_factor(); */
/* } */

InputMultisliceWrapper::FloatType InputMultisliceWrapper::set_incident_angle(const
    InputMultisliceWrapper::FloatType &theta) const {
  return handle_->set_incident_angle(theta);
}

InputMultisliceWrapper::FloatType InputMultisliceWrapper::get_phonon_rot_weight() const {
  return handle_->get_phonon_rot_weight();
}

void InputMultisliceWrapper::set_phi(const int &irot) {
  handle_->set_phi(irot);
}

InputMultisliceWrapper::FloatType InputMultisliceWrapper::get_propagator_factor(const
    InputMultisliceWrapper::FloatType &z) const {
  return handle_->get_propagator_factor(z);
}

InputMultisliceWrapper::FloatType InputMultisliceWrapper::Vr_factor() const {
  return handle_->Vr_factor();
}

InputMultisliceWrapper::FloatType InputMultisliceWrapper::gx_0() const {
  return handle_->gx_0();
}

InputMultisliceWrapper::FloatType InputMultisliceWrapper::gy_0() const {
  return handle_->gy_0();
}
  
int InputMultisliceWrapper::number_conf() {
  return handle_->number_conf();
}

int InputMultisliceWrapper::number_of_beams() {
  return handle_->number_of_beams();
}

bool InputMultisliceWrapper::is_multi_beam() {
  return handle_->is_multi_beam();
}
  
bool InputMultisliceWrapper::is_spec_rot_active() const {
  return handle_->is_spec_rot_active();
}

bool InputMultisliceWrapper::is_multislice() const {
  return handle_->is_multislice();
}

bool InputMultisliceWrapper::is_phase_object() const {
  return handle_->is_phase_object();
}

bool InputMultisliceWrapper::is_weak_phase_object() const {
  return handle_->is_weak_phase_object();
}

bool InputMultisliceWrapper::is_still_atom() const {
  return handle_->is_still_atom();
}

bool InputMultisliceWrapper::is_absorptive_model() const {
  return handle_->is_absorptive_model();
}

bool InputMultisliceWrapper::is_frozen_phonon() const {
  return handle_->is_frozen_phonon();
}

bool InputMultisliceWrapper::is_frozen_phonon_single_conf() const {
  return handle_->is_frozen_phonon_single_conf();
}

bool InputMultisliceWrapper::is_whole_spec() const {
  return handle_->is_whole_spec();
}

bool InputMultisliceWrapper::is_through_slices() const {
  return handle_->is_through_slices();
}

bool InputMultisliceWrapper::is_through_thick() const {
  return handle_->is_through_thick();
}

bool InputMultisliceWrapper::is_slicing_by_planes() const {
  return handle_->is_slicing_by_planes();
}

bool InputMultisliceWrapper::is_slicing_by_dz() const {
  return handle_->is_slicing_by_dz();
}

bool InputMultisliceWrapper::is_subslicing() const {
  return handle_->is_subslicing();
}

bool InputMultisliceWrapper::is_subslicing_whole_spec() const {
  return handle_->is_subslicing_whole_spec();
}

bool InputMultisliceWrapper::is_plane_wave() const {
  return handle_->is_plane_wave();
}

bool InputMultisliceWrapper::is_convergent_wave() const {
  return handle_->is_convergent_wave();
}

bool InputMultisliceWrapper::is_user_define_wave() const {
  return handle_->is_user_define_wave();
}

bool InputMultisliceWrapper::is_STEM() const {
  return handle_->is_STEM();
}

bool InputMultisliceWrapper::is_ISTEM() const {
  return handle_->is_ISTEM();
}

bool InputMultisliceWrapper::is_CBED() const {
  return handle_->is_CBED();
}

bool InputMultisliceWrapper::is_CBEI() const {
  return handle_->is_CBEI();
}

bool InputMultisliceWrapper::is_ED() const {
  return handle_->is_ED();
}

bool InputMultisliceWrapper::is_HRTEM() const {
  return handle_->is_HRTEM();
}

bool InputMultisliceWrapper::is_PED() const {
  return handle_->is_PED();
}

bool InputMultisliceWrapper::is_HCTEM() const {
  return handle_->is_HCTEM();
}

bool InputMultisliceWrapper::is_EWFS() const {
  return handle_->is_EWFS();
}

bool InputMultisliceWrapper::is_EWRS() const {
  return handle_->is_EWRS();
}

bool InputMultisliceWrapper::is_EWFS_SC() const {
  return handle_->is_EWFS_SC();
}

bool InputMultisliceWrapper::is_EWRS_SC() const {
  return handle_->is_EWRS_SC();
}

bool InputMultisliceWrapper::is_EELS() const {
  return handle_->is_EELS();
}

bool InputMultisliceWrapper::is_EFTEM() const {
  return handle_->is_EFTEM();
}

bool InputMultisliceWrapper::is_IWFS() const {
  return handle_->is_IWFS();
}

bool InputMultisliceWrapper::is_IWRS() const {
  return handle_->is_IWRS();
}

bool InputMultisliceWrapper::is_PPFS() const {
  return handle_->is_PPFS();
}

bool InputMultisliceWrapper::is_PPRS() const {
  return handle_->is_PPRS();
}

bool InputMultisliceWrapper::is_TFFS() const {
  return handle_->is_TFFS();
}

bool InputMultisliceWrapper::is_TFRS() const {
  return handle_->is_TFRS();
}

bool InputMultisliceWrapper::is_PropFS() const {
  return handle_->is_PropFS();
}

bool InputMultisliceWrapper::is_PropRS() const {
  return handle_->is_PropRS();
}

bool InputMultisliceWrapper::is_STEM_ISTEM() const {
  return handle_->is_STEM_ISTEM();
}

bool InputMultisliceWrapper::is_CBED_CBEI() const {
  return handle_->is_CBED_CBEI();
}

bool InputMultisliceWrapper::is_ED_HRTEM() const {
  return handle_->is_ED_HRTEM();
}

bool InputMultisliceWrapper::is_PED_HCTEM() const {
  return handle_->is_PED_HCTEM();
}

bool InputMultisliceWrapper::is_EWFS_EWRS() const {
  return handle_->is_EWFS_EWRS();
}

bool InputMultisliceWrapper::is_EWFS_EWRS_SC() const {
  return handle_->is_EWFS_EWRS_SC();
}

bool InputMultisliceWrapper::is_EELS_EFTEM() const {
  return handle_->is_EELS_EFTEM();
}

bool InputMultisliceWrapper::is_IWFS_IWRS() const {
  return handle_->is_IWFS_IWRS();
}

bool InputMultisliceWrapper::is_PPFS_PPRS() const {
  return handle_->is_PPFS_PPRS();
}

bool InputMultisliceWrapper::is_TFFS_TFRS() const {
  return handle_->is_TFFS_TFRS();
}

bool InputMultisliceWrapper::is_PropFS_PropRS() const {
  return handle_->is_PropFS_PropRS();
}

bool InputMultisliceWrapper::is_grid_FS() const {
  return handle_->is_grid_FS();
}

bool InputMultisliceWrapper::is_grid_RS() const {
  return handle_->is_grid_RS();
}

bool InputMultisliceWrapper::is_simulation_type_FS() const {
  return handle_->is_simulation_type_FS();
}

bool InputMultisliceWrapper::is_simulation_type_RS() const {
  return handle_->is_simulation_type_RS();
}

bool InputMultisliceWrapper::is_specimen_required() const {
  return handle_->is_specimen_required();
}

bool InputMultisliceWrapper::is_ISTEM_CBEI_HRTEM_HCTEM_EFTEM() const {
  return handle_->is_ISTEM_CBEI_HRTEM_HCTEM_EFTEM();
}

bool InputMultisliceWrapper::is_CBED_ED_EWFS_PED() const {
  return handle_->is_CBED_ED_EWFS_PED();
}

bool InputMultisliceWrapper::is_obj_lens_temp_spat() const {
  return handle_->is_obj_lens_temp_spat();
}

bool InputMultisliceWrapper::is_cond_lens_temp_spat() const {
  return handle_->is_cond_lens_temp_spat();
}

bool InputMultisliceWrapper::is_scanning() const {
  return handle_->is_scanning();
}

bool InputMultisliceWrapper::is_illu_mod_coherent() const {
  return handle_->is_illu_mod_coherent();
}

bool InputMultisliceWrapper::is_illu_mod_partial_coherent() const {
  return handle_->is_illu_mod_partial_coherent();
}

bool InputMultisliceWrapper::is_illu_mod_trans_cross_coef() const {
  return handle_->is_illu_mod_trans_cross_coef();
}

bool InputMultisliceWrapper::is_illu_mod_full_integration() const {
  return handle_->is_illu_mod_full_integration();
}

bool InputMultisliceWrapper::is_incoh_temporal_spatial() const {
  return handle_->is_incoh_temporal_spatial();
}

bool InputMultisliceWrapper::is_incoh_temporal() const {
  return handle_->is_incoh_temporal();
}

bool InputMultisliceWrapper::is_incoh_spatial() const {
  return handle_->is_incoh_spatial();
}

bool InputMultisliceWrapper::is_detector_circular() const {
  return handle_->is_detector_circular();
}

bool InputMultisliceWrapper::is_detector_radial() const {
  return handle_->is_detector_radial();
}

bool InputMultisliceWrapper::is_detector_matrix() const {
  return handle_->is_detector_matrix();
}

bool InputMultisliceWrapper::is_slice_storage() const {
  return handle_->is_slice_storage();
}

bool InputMultisliceWrapper::is_operation_mode_normal() const {
  return handle_->is_operation_mode_normal();
}

bool InputMultisliceWrapper::is_operation_mode_advanced() const {
  return handle_->is_operation_mode_advanced();
}

bool InputMultisliceWrapper::is_lvt_off() const {
  return handle_->is_lvt_off();
}

bool InputMultisliceWrapper::is_lvt_m() const {
  return handle_->is_lvt_m();
}

bool InputMultisliceWrapper::is_lvt_Cs3() const {
  return handle_->is_lvt_Cs3();
}

bool InputMultisliceWrapper::is_lvt_Cs5() const {
  return handle_->is_lvt_Cs5();
}

bool InputMultisliceWrapper::is_lvt_mfa2() const {
  return handle_->is_lvt_mfa2();
}

bool InputMultisliceWrapper::is_lvt_afa2() const {
  return handle_->is_lvt_afa2();
}

bool InputMultisliceWrapper::is_lvt_mfa3() const {
  return handle_->is_lvt_mfa3();
}

bool InputMultisliceWrapper::is_lvt_afa3() const {
  return handle_->is_lvt_afa3();
}

bool InputMultisliceWrapper::is_lvt_inner_aper_ang() const {
  return handle_->is_lvt_inner_aper_ang();
}

bool InputMultisliceWrapper::is_lvt_outer_aper_ang() const {
  return handle_->is_lvt_outer_aper_ang();
}






/* typedef float FloatType; */

/* void simulate( */
/*     mt::System_Configuration &system_conf, */
/*     mt::Input_Multislice<FloatType> &input_multislice, */ 
/*     mt::Output_Multislice<FloatType> &output_multislice) { */

/*   mt_run_multislice<FloatType, mt::e_host>( */
/*     system_conf, */
/*     input_multislice, */ 
/*     output_multislice); */
/* } */
